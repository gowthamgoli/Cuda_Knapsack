#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

#define index(i, j, N)  ((i)*(N+1)) + (j)

__device__ int maximum(int a, int b) { 
	return (a > b)? a : b; 
}

__global__ void knapsackKernel(int *profits, int *weights, int *f, int capacity, int i){

	int c = threadIdx.x;

	if(i==0 || c==0)	f[index(i,c,capacity)] = 0;
	else if(weights[i-1] <= c){
		f[index(i,c,capacity)] = maximum(f[index(i-1,c,capacity)], profits[i-1]+f[index(i-1,c-weights[i-1],capacity)]);
	}
	else
		f[index(i,c,capacity)] = f[index(i-1,c,capacity)];
}

void knapsackCuda(int *profits, int *weights, int c, int n, int *f){
	int *dev_profits, *dev_weights, *dev_f;

	hipMalloc((void**)&dev_f, (n+1)*(c+1)*sizeof(int));
	hipMalloc((void**)&dev_profits, n*sizeof(int));
	hipMalloc((void**)&dev_weights, n*sizeof(int));

	hipMemcpy(dev_profits, profits, n*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_weights, weights, n*sizeof(int), hipMemcpyHostToDevice);

	int i=0;
	while(i<=n){
		knapsackKernel<<<1, c+1>>>(dev_profits, dev_weights, dev_f, c, i);
		i++;
	}

	hipMemcpy(f, dev_f, (n+1)*(c+1)*sizeof(int), hipMemcpyDeviceToHost);
	
	hipFree(dev_profits);
	hipFree(dev_weights);
	hipFree(dev_f);
}

int main() {
	int i;
	int n = 3;
	int *profits = (int*)malloc(n*sizeof(int));
	int *weights = (int*)malloc(n*sizeof(int));

	FILE *myFile;
    myFile = fopen("rand.txt", "r");

	for (i = 0; i < n; i++)
    {
        fscanf(myFile, "%d %d", &profits[i], &weights[i]);
    }

    int capacity = 5;

    //int n = sizeof(profits)/sizeof(int);

    int *f = (int *)malloc((n+1)*(capacity+1)*sizeof(int));

    knapsackCuda(profits, weights, capacity, n, f);

    int c;
	for(i=0; i<=n ; i++){
		for(c=0; c<=capacity; c++){
			printf("%d ", f[index(i,c,capacity)]);
		}
		printf("\n");
	}

}
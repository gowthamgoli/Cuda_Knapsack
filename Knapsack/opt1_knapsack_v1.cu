#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

#define TILE_WIDTH 512
#define index(i, j, N)  ((i)*(N+1)) + (j)

int maximum(int a, int b) { 
	return (a > b)? a : b; 
}

__global__ void knapsackKernel(int *profits, int *weights, int *input_f, int *output_f, int capacity, int c_min, int k){

	int c = blockIdx.x*512 + threadIdx.x;
	if(c<c_min || c>capacity){return;}
	if(input_f[c] < input_f[c-weights[k-1]]+profits[k-1]){
		output_f[c] = input_f[c-weights[k-1]]+profits[k-1];
	}
	else{
		output_f[c] = input_f[c];
	}
}

void knapsackCuda(int *profits, int *weights, int capacity, int n, int *f0, int *f1){
	int *dev_profits, *dev_weights, *dev_f0, *dev_f1;
	int sumW = 0;
	int i,c; 

	for(i=0; i<n; i++){
		sumW = sumW + weights[i];
	}

	hipMalloc((void**)&dev_f0, (capacity+1)*sizeof(int));
	hipMalloc((void**)&dev_f1, (capacity+1)*sizeof(int));
	hipMalloc((void**)&dev_profits, n*sizeof(int));
	hipMalloc((void**)&dev_weights, n*sizeof(int));

	hipMemcpy(dev_profits, profits, n*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_weights, weights, n*sizeof(int), hipMemcpyHostToDevice);
	hipMemset(dev_f0, 0, (capacity+1)*sizeof(int));
	hipMemset(dev_f1, 0, (capacity+1)*sizeof(int));

	/*int p;
	for(p=0; p<=capacity; p++)	printf("%d ", dev_f1[p]);
	printf("\n");*/

	int k=1;
	while(k<=n){
		sumW = sumW - weights[k-1];
		c = maximum(capacity-sumW, weights[k-1]);

		//printf("k = %d\n", k);
		//printf("%d\n", c);
		
		dim3 dimGrid(ceil(1.0*(capacity-0+1)/TILE_WIDTH), 1, 1);
		dim3 dimBlock(TILE_WIDTH,1,1);

		if(k%2==0){
			hipMemcpy(dev_f1, dev_f0, (capacity+1)*sizeof(int), hipMemcpyDeviceToDevice);
			knapsackKernel<<<dimGrid, dimBlock>>>(dev_profits, dev_weights, dev_f0, dev_f1, capacity, c, k);
			//cudaDeviceSynchronize();
			/*cudaMemcpy(f1, dev_f1, (capacity+1)*sizeof(int), cudaMemcpyDeviceToHost);
			int p;
			for(p=0; p<=capacity; p++)	printf("%d ", f1[p]);
			printf("\n");*/
		}
		else{
			hipMemcpy(dev_f0, dev_f1, (capacity+1)*sizeof(int), hipMemcpyDeviceToDevice);
			knapsackKernel<<<dimGrid, dimBlock>>>(dev_profits, dev_weights, dev_f1, dev_f0, capacity, c, k);	
			//cudaDeviceSynchronize();
			/*cudaMemcpy(f0, dev_f0, (capacity+1)*sizeof(int), cudaMemcpyDeviceToHost);
			int p;
			for(p=0; p<=capacity; p++)	printf("%d ", f0[p]);
			printf("\n");*/
		}
		k++;
	}

	hipMemcpy(f0, dev_f0, (capacity+1)*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(f1, dev_f1, (capacity+1)*sizeof(int), hipMemcpyDeviceToHost);
	
	hipFree(dev_profits);
	hipFree(dev_weights);
	hipFree(dev_f0);
	hipFree(dev_f1);
}

int main() {
    int i;
	int n = 10000;
	int *profits = (int*)malloc(n*sizeof(int));
	int *weights = (int*)malloc(n*sizeof(int));

	FILE *myFile;
    myFile = fopen("rand.txt", "r");

	for (i = 0; i < n; i++)
    {
        fscanf(myFile, "%d %d", &profits[i], &weights[i]);
    }

    int capacity = 0;
	for(i=0; i<n; i++){
		capacity = capacity + weights[i];
	}
	capacity = capacity/2;
	//capacity = 1000;

	printf("capacity = %d\n", capacity);

    
    int *f0 = (int *)malloc((capacity+1)*sizeof(int));
    int *f1 = (int *)malloc((capacity+1)*sizeof(int));

    knapsackCuda(profits, weights, capacity, n, f0, f1);

    if(n%2==0){
    	//int p;
		//for(p=0; p<=capacity; p++){	printf("%d ", f1[p]);}
    	printf("%d\n", f1[capacity]);
    }
    else{
    	//int p;
		//for(p=0; p<=capacity; p++)	{printf("%d ", f0[p]);}
    	printf("%d\n", f0[capacity]);
    }
}

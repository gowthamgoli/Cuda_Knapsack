/*
 *  Please write your name and net ID below
 *
 *  Last name: Bora
 *  First name: Anuj
 *  Net ID: aab688
 *
 */


/*
 * This file contains the code for doing the heat distribution problem.
 * You do not need to modify anything except starting  gpu_heat_dist() at the bottom
 * of this file.
 * In gpu_heat_dist() you can organize your data structure and the call to your
 * kernel(s) that you need to write too.
 *
 * You compile with:
 * 		nvcc -o heatdist heatdist.cu
 */

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>

/* To index element (i,j) of a 2D array stored as 1D */
#define index(i, j, N)  ((i)*(N)) + (j)
#define BLOCKSIZE 256


/*****************************************************************/

// Function declarations: Feel free to add any functions you want.
int  gpu_levenshtein( char * ,
                       char * ,
                     int , int );
void printMatrix(int*, unsigned int);
int getBlocks(int , int );

/*****************************************************************/

/*****************************************************************/
__global__ void warmUpGPU()
{
  // do nothing
}

__global__ void antiparallelUT(unsigned short* d_A, int step, int len1, int len2,
  const char* d_word1, const char* d_word2, int* d_result)
{

	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = step-i;

  if (i == 0 && j <= len2) {
    d_A[index(i + j, j, len2 + 1)] = j;
  }

  if (j == 0 && i <= len2) {
    d_A[index(i + j, j, len2 + 1)] = i;
  }

  if (i <= len1 && j <= len2 && i>=1 && j>=1) {
      //printf("[%d][%d]\n", i, j);
      //d_A[i*N+j] = 1;

      int delete_count;
      int insert;
      int substitute;

      char c2;
      char c1;

      c1 = d_word1[i-1];
      c2 = d_word2[j-1];

      int score = 1;


      if (c1 == c2) {
          score = 0;
      }
      /*
      delete_count = d_A[index(i - 1, j, len2 + 1)] + 1;
      insert = d_A[index(i, j - 1, len2 + 1)] + 1;
      substitute = d_A[index(i - 1, j - 1, len2 + 1)] + score;
      */

      int old_x = i - 1;
      int y = j;
      int x = old_x + y;
      if (x > len1 ) {
        x = x - len1 - 1;
      }
      delete_count = d_A[index(x, y, len2 + 1)] + 1;

      old_x = i - 1;
      y = j - 1;
      x = old_x + y;
      if (x > len1 ) {
        x = x - len1 - 1;
      }
      insert = d_A[index(x, y, len2 + 1)] + score;

      old_x = i;
      y = j - 1;
      x = old_x + y;
      if (x > len1 ) {
        x = x - len1 - 1;
      }
      substitute = d_A[index(x, y, len2 + 1)] + 1;


      int min;

      if (delete_count < insert) {
        min = delete_count;
      } else {
         min = insert;
      }

      if (substitute < min) {
        min = substitute;
      }
      x = i + j;
      y = j;
      if (x > len1 ) {
        x = x - len1 - 1;
      }
      d_A[index(x, y, len2 + 1)] = min;

      if (i == len1 && j == len2) {
        //printf("updating result...");
        //printf("\nmin = %d\n", min);
         d_result[0] = min;
      }

  }

  //__syncthreads();
}


// Print
void printMatrix(int* playground, int len1, int len2)
{

  for (int i = 0; i < len1; i++)
  {
    for (int j = 0; j < len2; j++)
    {
      printf("%d ", playground[index(i,j,len2)]);
    }
    printf("\n ");
  }

}

int getBlocks(int a, int b) {
  return (a % b != 0) ? (a / b + 1) : (a / b);
}

/*****************************************************************/

int main(int argc, char * argv[])
{
  //char string1[4] = {'a', 'n', 'u', 'j'};
  //char string2[5] = {'a', 'n', 't', 'j', 'b'};

  int size =32000;
  char string1[size];
  char string2[size];
  for (int i = 0; i < size; i++) {
    string1[i] = 'a';
    string2[i] = 'b';
  }

  //string1[0] = 'b';

  int dis = gpu_levenshtein(string1, string2, size, size);

  printf("Result = %d", dis);

  return 0;

}

/***************** The GPU version: Write your code here *********************/
int  gpu_levenshtein( char * word1,
                     char * word2,
                     int len1, int len2)
{
  warmUpGPU<<<1, 1>>>();

  int N = len1 + 1;
  int size = (N) * (N) * sizeof(unsigned short);
  int word1_size = len1 * sizeof(char);
  int word2_size = len2 * sizeof(char);
  int* result_host = (int*)malloc(sizeof(int));
  result_host[0] = 0;

  unsigned short *zero_matrix;
  unsigned short *d_A;
  int *d_result;
  char *d_word1, *d_word2;

  zero_matrix = (unsigned short *)calloc((N) * (N), sizeof(unsigned short));

  if(zero_matrix == NULL)
  {
    printf("Memory allocation failed on CPU");
  }
//  int i, j;
  /*
  for(i = 0; i <= len1; i++) {
    for (j = 0; j <= len2; j++) {
      zero_matrix[index(i, j, (len2 + 1))] = 0;
    }
  } */
  /*
  for (i = 0; i < (2 * N)-1; i++) {
    for (j = 0; j < N; j++) {
      zero_matrix[index(i, 0, (len2 + 1))] = 0;
      if (i == 0) {
        zero_matrix[index(0, j, (len2 + 1))] = j;
      }
      if (j == 0) {
        zero_matrix[index(i, 0, (len2 + 1))] = i;
      }
    }
  }*/

  //printMatrix(zero_matrix, ((2 * N)-1), N);
  /*
  for (i = 0; i <= len1; i++) {
      zero_matrix[index(i, 0, (len2 + 1))] = i;
  }
  for (i = 0; i <= len2; i++) {
      zero_matrix[index(0, i, (len2 + 1))] = i;
  }
  */

  if ( hipSuccess != hipMalloc((void **) &d_A, size) )
  {
      printf( "Error in allocating memory on GPU!!\n" );
  }

  hipMalloc((void **) &d_result, sizeof(int));
  hipMalloc((void **) &d_word1, word1_size);
  hipMalloc((void **) &d_word2, word2_size);

  hipMemcpy(d_A, zero_matrix, size, hipMemcpyHostToDevice);
  hipMemcpy(d_result, result_host, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_word1, word1, word1_size, hipMemcpyHostToDevice);
  hipMemcpy(d_word2, word2, word2_size, hipMemcpyHostToDevice);
  int step = 0;

  for ( step = 0; step < (2 * N) - 1; step++) {

    dim3 dimBlock(BLOCKSIZE);
    dim3 dimGrid(getBlocks(step, BLOCKSIZE));

    if (step == 0) {
      antiparallelUT<<<1,dimBlock.x>>>(d_A,step,len1, len2, d_word1, d_word2, d_result);
    } else {
      antiparallelUT<<<dimGrid.x,dimBlock.x>>>(d_A,step,len1, len2, d_word1, d_word2, d_result);
    }
  }


  // Step 3 : Bring result back to host
  //cudaMemcpy(zero_matrix, d_A, size, cudaMemcpyDeviceToHost);
  //int *result_host = 0;
  hipMemcpy(result_host, d_result, sizeof(int), hipMemcpyDeviceToHost);

  // Step 4 : Free device memory
  hipFree(d_A);
  hipFree(d_word1);
  hipFree(d_word2);
  //printf("\n\n\n");
  //printMatrix(zero_matrix, N, N);
  //printMatrix(flags, len1, len2);
  //printf("%d \n", index(len1, len2, len2 + 1));
  //int result = zero_matrix[index(len1, len2, (len2 + 1))];
  free(zero_matrix);
  return result_host[0];
}
